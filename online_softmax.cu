#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// online softmax kernel 
__global__ void online_softmax_kernel(float* input, float* output, int n){
    // Use shared memory to store running max and sum
    extern __shared__ float shared_data[];
    float* running_max = &shared_data[0];      // stores current max
    float* running_sum = &shared_data[1];      // stores current sum
    float* temp_outputs = &shared_data[2];     // remaining space for temporary outputs
    
    // Initialize shared memory (only thread 0)
    if (threadIdx.x == 0) {
        *running_max = -INFINITY;
        *running_sum = 0.0f;
    }
    __syncthreads();
    
    // Process elements sequentially - this is the key to online softmax
    // Each thread processes elements in order, updating running statistics
    for (int i = 0; i < n; i++) {
        // Only one thread at a time processes an element
        if (threadIdx.x == 0) {
            float x_i = input[i];
            float old_max = *running_max;
            
            if (x_i > old_max) {
                // New maximum found - rescale previous sum and update max
                *running_sum = *running_sum * expf(old_max - x_i) + 1.0f;
                *running_max = x_i;
            } else {
                // Current element is not new max - just add to sum
                *running_sum = *running_sum + expf(x_i - *running_max);
            }
        }
        __syncthreads();
    }
    
    // compute final softmax values using the final max and sum
    int thread_id = threadIdx.x;
    if (thread_id < n) {
        output[thread_id] = expf(input[thread_id] - *running_max) / *running_sum;
    }
}


// Utility function to print array
void print_array(const char* name, float* arr, int n) {
    printf("%s: [", name);
    for (int i = 0; i < n; i++) {
        printf("%.6f", arr[i]);
        if (i < n - 1) printf(", ");
    }
    printf("]\n");
}



int main() {
    // Test parameters
    const int n = 8;  // Size of input vector (must be <= block size for this online implementation)
    const int block_size = 256;  // CUDA block size
    
    printf("Online Softmax CUDA Implementation\n");
    printf("Input size: %d\n\n", n);
    
    // Allocate host memory
    float* h_input = (float*)malloc(n * sizeof(float));
    float* h_output_gpu = (float*)malloc(n * sizeof(float));
    
    // Initialize input with sample data
    printf("Initializing input data...\n");
    for (int i = 0; i < n; i++) {
        h_input[i] = (float)(rand() % 100) / 10.0f - 5.0f;  // Random values between -5 and 5
    }
    
    print_array("Input", h_input, n);
    // Allocate device memory
    float* d_input;
    float* d_output;
    CUDA_CHECK(hipMalloc(&d_input, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_output, n * sizeof(float)));
    // Copy input data to device
    CUDA_CHECK(hipMemcpy(d_input, h_input, n * sizeof(float), hipMemcpyHostToDevice));
    // Launch kernel
    printf("\nLaunching CUDA kernel...\n");
    dim3 grid(1);  // Single block for this online implementation
    dim3 block(block_size);
    size_t shared_mem_size = block_size * sizeof(float);
    // Create CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    // Record start time
    CUDA_CHECK(hipEventRecord(start));
    // Launch kernel
    online_softmax_kernel<<<grid, block, shared_mem_size>>>(d_input, d_output, n);
    // Record end time
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());
    // Calculate elapsed time
    float gpu_time;
    CUDA_CHECK(hipEventElapsedTime(&gpu_time, start, stop));
    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_output_gpu, d_output, n * sizeof(float), hipMemcpyDeviceToHost));
    // Print results
    print_array("GPU Output", h_output_gpu, n);    
    // Print timing
    printf("GPU execution time: %.3f ms\n", gpu_time);
    
    // Verify that output sums to 1 (property of softmax)
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
        sum += h_output_gpu[i];
    }
    printf("Sum of GPU output: %.6f (should be ~1.0)\n", sum);
    
    // Cleanup
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    free(h_input);
    free(h_output_gpu);
    
    printf("\nProgram completed successfully!\n");
    return 0;
}

/*
  
 * To run:
 *   ./online_softmax
 */
